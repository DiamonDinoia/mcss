#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include "common.h"
#include "mcss_gpu.h"

namespace Gpu {

static constexpr size_t div_rounding_up(size_t dividend, size_t divisor) {
    return (dividend + divisor - 1) / divisor;
}

__global__ static void compute_terminal_positions(
    const unsigned seed, real_type *longHist, real_type *transHist,
    real_type screening_param, real_type mean_free_path, real_type track_limit,
    const unsigned int longHistNumBins, const unsigned int transHistNumBins,
    real_type longDistInvD, real_type transDistInvD, int numSims,
    int thread_histories) {
    int threadX = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadX;

    // Initialise local histograms
    __shared__ real_type shared_longHist[longiDistNumBin];
    __shared__ real_type shared_transHist[transDistNumBin];

    hiprandState local_state;
    hiprand_init(i, seed, 0, &local_state);

    if (threadX < longHistNumBins) {
        shared_longHist[threadX] = 0.0;
    }
    if (threadX < transHistNumBins) {
        shared_transHist[threadX] = 0.0;
    }

    // Ensure that every thread has initialised the histograms
    __syncthreads();

    // Initialise local variables
    real_type trackPosition_x;
    real_type trackPosition_y;
    real_type trackPosition_z;
    real_type trackDirection_x;
    real_type trackDirection_y;
    real_type trackDirection_z;
    real_type stepLength;
    real_type trackLength;
    bool stop;
    unsigned int local_idx;
    real_type rand_val;

    // Perform multiple simulations per thread
    for (int j = 0; j < thread_histories; j++) {
        // Use strides of size blockDim.x*gridDim.x to divide work as
        // equally as possible
        local_idx =
            j * blockDim.x * gridDim.x + blockDim.x * blockIdx.x + threadX;

        if (local_idx >= numSims) {
            break;
        }

        // Initialise local variables
        trackPosition_x = 0.0;
        trackPosition_y = 0.0;
        trackPosition_z = 0.0;
        trackDirection_x = 0.0;
        trackDirection_y = 0.0;
        trackDirection_z = 1.0;

        stepLength = 0.0;
        trackLength = 0.0;
        stop = false;

        // Start the simulation
        while (!stop) {
            // Compute step length
            rand_val = hiprand_uniform(&local_state);

            stepLength = -mean_free_path * log(rand_val);
            if (trackLength > track_limit) {  // Last step, so shorten it
                                              // and stop afterwards
                stepLength = track_limit - trackLength;
                stop = true;
            }

            // Update track positions
            trackPosition_x += stepLength * trackDirection_x;
            trackPosition_y += stepLength * trackDirection_y;
            trackPosition_z += stepLength * trackDirection_z;
            trackLength += stepLength;

            // Update track direction if we are not at the end already
            if (!stop) {
                // Compute new positions based on random collision direction
                rand_val = hiprand_uniform(&local_state);

                real_type cost = sampleCosTheta(screening_param, rand_val);
                real_type sint = sqrt((1.0 - cost) * (1.0 + cost));
                rand_val = hiprand_uniform(&local_state);
                real_type phi = 2.0 * kPI * rand_val;

                real_type u1 = sint * cos(phi);
                real_type u2 = sint * sin(phi);
                real_type u3 = cost;

                // Rotate back to lab frame
                rotateToLabFrame(u1, u2, u3, trackDirection_x, trackDirection_y,
                                 trackDirection_z);

                trackDirection_x = u1;
                trackDirection_y = u2;
                trackDirection_z = u3;
            }
        }

        // Compute longitudinal deviation and its bin index
        real_type longitudinal_deviation = trackPosition_z / trackLength;
        unsigned int longIdx =
            (unsigned int)((longitudinal_deviation + 1.0) * longDistInvD);

        // Compute transversal deviation and its bin index
        real_type transversal_deviation =
            sqrt(trackPosition_x * trackPosition_x +
                 trackPosition_y * trackPosition_y) /
            trackLength;
        unsigned int transIdx =
            (unsigned int)(transversal_deviation * transDistInvD);

        // Write simulation result to histograms
        atomicAdd(&(shared_longHist[longIdx]), longDistInvD / numSims);
        atomicAdd(&(shared_transHist[transIdx]), transDistInvD / numSims);
    }

    // Sync up threads after simulations
    __syncthreads();

    // Add shared histogram to global histogram
    if (threadX < longHistNumBins) {
        atomicAdd(&(longHist[threadX]), shared_longHist[threadX]);
    }
    if (threadX < transHistNumBins) {
        atomicAdd(&(transHist[threadX]), shared_transHist[threadX]);
    }
}

__global__ void initialise_RNG(hiprandState *states, int numStates) {
    unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long seed = 0;

    // Use the same seed but different sequence number
    if (i < numStates) {
        hiprand_init(i, seed, 0, &states[i]);
    }
}

/*
 * Host code
 */

Histograms Simulate(Material material, int numHists) {
    real_type *longHist_h, *transHist_h;
    hipError_t cuda_ret;
    const auto seed = 42;
    // Wake up GPU (seems to be a problem with Peregrine GPUs being in
    // sleep mode, so I wake them up before timing the application)
    // hipFree(0);
    // hipDeviceSynchronize();

    // Initialise space on host for histograms
    longHist_h = (real_type *)malloc(longiDistNumBin * sizeof(real_type));
    transHist_h = (real_type *)malloc(transDistNumBin * sizeof(real_type));

    // Initialise constants
    const real_type theScrPar = computeScrParam(material, thePC2);
    const real_type theMFP = computeMFP(material, theBeta2, theScrPar);
    const real_type theLimit = theMFP * 33.5;

    // Define block and grid dimensions
    constexpr auto num_threads = NUM_THREADS;
    auto num_blocks = div_rounding_up(numHists, num_threads);
    // CUDA has a limit on the grid size
    num_blocks = math::min(num_blocks, std::numeric_limits<unsigned int>::max());
    // Since we are using __shared__ memory we need to check we do not exceed
    // 48KB
    const auto shared_mem_size = 48000;  // bytes
    const auto shared_mem_usage =
        sizeof(real_type) * (longiDistNumBin + transDistNumBin);
    num_blocks = math::min(shared_mem_size / shared_mem_usage, num_blocks);
    auto thread_histories = div_rounding_up(numHists, num_blocks*num_threads);
    thread_histories = math::max(thread_histories, 1);
    // Initialise histograms on GPU
    real_type *longHist_d;
    real_type *transHist_d;
    cuda_ret = hipMalloc(&longHist_d, longiDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipMalloc(&transHist_d, transDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise transHist on the GPU.\n");
        exit(-1);
    }

    // Set histogram values to 0
    cuda_ret = hipMemset(longHist_d, 0, longiDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set longHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipMemset(transHist_d, 0, transDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set transHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    // Run simulation on GPU
    compute_terminal_positions<<<num_blocks, num_threads>>>(
        seed, longHist_d, transHist_d, theScrPar, theMFP, theLimit,
        longiDistNumBin, transDistNumBin, longiDistInvD, transDistInvD,
        numHists, thread_histories);

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to complete simulation kernel.\n");
        exit(-1);
    }

    // Retrieve histograms from GPU
    cuda_ret =
        hipMemcpy(longHist_h, longHist_d, longiDistNumBin * sizeof(real_type),
                   hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve longitudinal histogram from GPU.\n");
        exit(-1);
    }
    cuda_ret =
        hipMemcpy(transHist_h, transHist_d,
                   transDistNumBin * sizeof(real_type), hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    // Copy array pointers into histogram variable
    Histograms histograms{longiDistNumBin, transDistNumBin};
    memcpy(&(histograms.longiHist[0]), &(longHist_h[0]),
           longiDistNumBin * sizeof(real_type));
    memcpy(&(histograms.transHist[0]), &(transHist_h[0]),
           transDistNumBin * sizeof(real_type));

    // Free buffers
    hipFree(longHist_d);
    hipFree(transHist_d);

    free(longHist_h);
    free(transHist_h);

    return histograms;
}

Histograms Simulate() { return Simulate(GOLD, 1000000); }

}  // namespace Gpu
