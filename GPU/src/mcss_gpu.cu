#include "hip/hip_runtime.h"
// M. Novak, 9 October 2020 (NOTE: energy is in [MeV] and length in [mm].)
// -----------------------------------------------------------------------
// Modified by R. Brandt 2022
// Modified by M. Barbone 2022
#include <VectorXoshiro/xoshiroPlusPlus.h>

#include <cmath>
#include <iostream>
#include <memory>
#include <random>

#include "common.h"
#include "mcss_gpu.h"
#include "mixmax/mixmax.h"

namespace Gpu {

static constexpr size_t div_rounding_up(size_t dividend, size_t divisor) {
    return (dividend + divisor - 1) / divisor;
}

template <typename T>
__device__ __attribute__((noinline)) static constexpr auto getGenerator(
    std::uint64_t i, std::uint64_t seed) {
    if constexpr (std::is_same_v<T, hiprandStatePhilox4_32_10_t> ||
                  std::is_same_v<T, hiprandStateMRG32k3a_t>) {
        T local_state{};
        hiprand_init(i, seed, 0, &local_state);
        return local_state;
    } else
        return T(seed, i);
}

template <typename T>
__device__ inline auto getUniform(T& rng
#ifdef COUNT_NUMBERS
                                  ,
                                  std::uint64_t* counter
#endif
) {
#ifdef COUNT_NUMBERS
    counter[blockIdx.x * blockDim.x + threadIdx.x]++;
#endif
    if constexpr (std::is_same_v<T, hiprandStatePhilox4_32_10_t> ||
                  std::is_same_v<T, hiprandStateMRG32k3a_t>) {
        return hiprand_uniform(&rng);
    }
    if constexpr (std::is_same_v<T, XoshiroPlusPlus>) { return rng.uniform(); }
    if constexpr (std::is_same_v<T, MIXMAX::MixMaxRng8> ||
                  std::is_same_v<T, CounterRNG> || std::is_same_v<T, ChaCha8>) {
        return rng.Uniform();
    }
}

template <typename T>
__global__ static void compute_terminal_positions(
    unsigned int seed, unsigned int* longHist, unsigned int* transHist,
    const real_type theScrPar, const real_type theMFP, const real_type theLimit,
    const int numSims, const int thread_histories
#ifdef COUNT_NUMBERS
    ,
    std::uint64_t* counter
#endif
) {
    const auto threadX = threadIdx.x;
    const auto i       = blockIdx.x * blockDim.x + threadX;

    // Load cuda random state into local register memory
    auto local_state = getGenerator<T>(i, seed);
#ifdef USE_SHARED_MEMORY
    __shared__ unsigned int shared_longHist[longiDistNumBin];
    __shared__ unsigned int shared_transHist[transDistNumBin];
    if (threadX < longiDistNumBin) { shared_longHist[threadX] = 0.0; }
    if (threadX < transDistNumBin) { shared_transHist[threadX] = 0.0; }

    // Ensure that every thread has initialised the histograms
    __syncthreads();
#endif
    // Perform multiple simulations per thread
    for (int j = 0; j < thread_histories; j++) {
        // Use strides of size blockDim.x*gridDim.x to divide work as
        // equally as possible

        const auto local_idx =
            j * blockDim.x * gridDim.x + blockDim.x * blockIdx.x + threadX;

        if (local_idx >= numSims) { break; }

        Track     aTrack{};
        real_type trackLength = 0.0;
        bool      stop        = false;
        do {
            real_type stepLength = -theMFP * std::log(getUniform(local_state
#ifdef COUNT_NUMBERS
                                                                 ,
                                                                 counter
#endif
                                                                 ));
            trackLength += stepLength;
            if (trackLength > theLimit) {
                stepLength = theLimit - aTrack.fTrackLength;
                stop       = true;
            }
            aTrack.fPosition[0] += aTrack.fDirection[0] * stepLength;
            aTrack.fPosition[1] += aTrack.fDirection[1] * stepLength;
            aTrack.fPosition[2] += aTrack.fDirection[2] * stepLength;
            aTrack.fTrackLength += stepLength;
            if (!stop) {
                const real_type cost =
                    sampleCosTheta(theScrPar, getUniform(local_state
#ifdef COUNT_NUMBERS
                                                         ,
                                                         counter
#endif
                                                         ));
                const real_type dum0 = 1.0 - cost;
                const real_type sint = std::sqrt(dum0 * (2.0 - dum0));
                const real_type phi  = 2.0 * kPI *
                                      getUniform(local_state
#ifdef COUNT_NUMBERS
                                                 ,
                                                 counter
#endif
                                      );
                real_type u1 = sint * std::cos(phi);
                real_type u2 = sint * std::sin(phi);
                real_type u3 = cost;
                rotateToLabFrame(u1, u2, u3, aTrack.fDirection[0],
                                 aTrack.fDirection[1], aTrack.fDirection[2]);
                aTrack.fDirection[0] = u1;
                aTrack.fDirection[1] = u2;
                aTrack.fDirection[2] = u3;
            }
        } while (!stop);

        // Calculate final longitudinal location.
        const real_type longi = aTrack.fPosition[2] / aTrack.fTrackLength;
        const unsigned  lIndx = (longi + 1.0) * longiDistInvD;
        // Increment the relevant element of the thread-specific
        // longitudinal histogram.
        const real_type trans =
            std::sqrt(aTrack.fPosition[0] * aTrack.fPosition[0] +
                      aTrack.fPosition[1] * aTrack.fPosition[1]) /
            aTrack.fTrackLength;
        const unsigned tIndx = trans * transDistInvD;
        // Increment the relevent element of the thread-specific
        // trans histogram.
#ifndef USE_SHARED_MEMORY
        atomicAdd(&(longHist[lIndx]), 1);
        atomicAdd(&(transHist[tIndx]), 1);
#endif
#ifdef USE_SHARED_MEMORY

        atomicAdd(&(shared_longHist[lIndx]), 1);
        atomicAdd(&(shared_transHist[tIndx]), 1);
#endif
    }
#ifdef USE_SHARED_MEMORY
    // Add shared histogram to global histogram
    if (threadX < longiDistNumBin) {
        atomicAdd(&(longHist[threadX]), shared_longHist[threadX]);
    }
    if (threadX < transDistNumBin) {
        atomicAdd(&(longHist[threadX]), shared_transHist[threadX]);
    }
#endif
}

/*
 * Host code
 */
template <typename T>
Histograms Simulate(Material material, int numHists) {
    hipError_t cuda_ret;
    const auto  seed = 42;

#ifndef USE_SHARED_MEMORY
    cuda_ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_terminal_positions<T>),
                                      hipFuncCachePreferL1);
#else
    cuda_ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_terminal_positions<T>),
                                      hipFuncCachePreferShared);
#endif
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    // Initialise space on host for histograms
    auto longHist_h  = std::make_unique<unsigned int[]>(longiDistNumBin);
    auto transHist_h = std::make_unique<unsigned int[]>(transDistNumBin);
    // Initialise constants
    const real_type theScrPar = computeScrParam(material, thePC2);
    const real_type theMFP    = computeMFP(material, theBeta2, theScrPar);
    const real_type theLimit  = theMFP * 33.5;

    // Define block and grid dimensions
    constexpr auto num_threads = NUM_THREADS;
    auto           num_blocks  = div_rounding_up(numHists, num_threads);
    // CUDA has a limit on the grid size
    num_blocks            = math::min(num_blocks, GRID_SIZE);

    auto thread_histories = div_rounding_up(numHists, num_blocks * num_threads);
    thread_histories      = math::max(thread_histories, 1);

#ifdef COUNT_NUMBERS
    // count the random numbers generated
    const auto numRands       = num_threads * num_blocks;
    auto       random_numbers = std::make_unique<std::uint64_t[]>(numRands);
    decltype(random_numbers.get()) random_numbers_d;

    cuda_ret = hipMalloc(&random_numbers_d, numRands * sizeof(std::uint64_t));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise rng counter on the GPU.\n");
        exit(-1);
    }

    cuda_ret =
        hipMemset(random_numbers_d, 0, numRands * sizeof(std::uint64_t));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set rng counter to zero on the GPU.\n");
        exit(-1);
    }
#endif
    // Initialise histograms on GPU
    unsigned int* longHist_d;
    unsigned int* transHist_d;
    cuda_ret = hipMalloc(&longHist_d, longiDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipMalloc(&transHist_d, transDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise transHist on the GPU.\n");
        exit(-1);
    }

    // Set histogram values to 0
    cuda_ret =
        hipMemset(longHist_d, 0, longiDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set longHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret =
        hipMemset(transHist_d, 0, transDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set transHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    // Run simulation on GPU
    compute_terminal_positions<T><<<num_blocks, num_threads>>>(
        seed, longHist_d, transHist_d, theScrPar, theMFP, theLimit, numHists,
        thread_histories
#ifdef COUNT_NUMBERS
        ,
        random_numbers_d
#endif
    );

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to complete simulation kernel.\n");
        exit(-1);
    }

#ifdef COUNT_NUMBERS
    // Retrieve random numbers from GPU
    cuda_ret =
        hipMemcpy(random_numbers.get(), random_numbers_d,
                   numRands * sizeof(std::uint64_t), hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve random numbers from GPU.\n");
        exit(-1);
    }
#endif

    // Retrieve histograms from GPU
    cuda_ret = hipMemcpy(longHist_h.get(), longHist_d,
                          longiDistNumBin * sizeof(unsigned int),
                          hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve longitudinal histogram from GPU.\n");
        exit(-1);
    }
    cuda_ret = hipMemcpy(transHist_h.get(), transHist_d,
                          transDistNumBin * sizeof(unsigned int),
                          hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }
#ifdef COUNT_NUMBERS
    // sum the random numbers
    const std::uint64_t sum = std::accumulate(
        random_numbers.get(), random_numbers.get() + numRands, 0UL);
    std::cout << "Random numbers generated: " << sum << std::endl;
#endif
    // Copy array pointers into histogram variable
    Histograms histograms{longiDistNumBin, transDistNumBin};
    for (int i = 0; i < longiDistNumBin; ++i) {
        histograms.longiHist[i] = longHist_h[i] * (longiDistInvD / numHists);
    }
    for (int i = 0; i < transDistNumBin; ++i) {
        histograms.transHist[i] = transHist_h[i] * (transDistInvD / numHists);
    }

    // Free buffers
#ifdef COUNT_NUMBERS
    hipFree(random_numbers_d);
#endif
    hipFree(longHist_d);
    hipFree(transHist_d);
    return histograms;
}

template Histograms Simulate<XoshiroPlusPlus>(Material, int);
template Histograms Simulate<MIXMAX::MixMaxRng8>(Material, int);
template Histograms Simulate<hiprandStatePhilox4_32_10>(Material, int);
template Histograms Simulate<hiprandStateMRG32k3a>(Material, int);
template Histograms Simulate<CounterRNG>(Material, int);
template Histograms Simulate<ChaCha8>(Material, int);

}  // namespace Gpu
