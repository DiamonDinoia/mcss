#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <random>

#include "common.h"
#include "mcss_gpu.h"

namespace Gpu {

static constexpr size_t div_rounding_up(size_t dividend, size_t divisor) {
    return (dividend + divisor - 1) / divisor;
}

__global__ static void compute_terminal_positions(
    const unsigned seed, unsigned int* longHist, unsigned int* transHist,
    real_type screening_param, real_type mean_free_path, real_type track_limit,
    const unsigned int longHistNumBins, const unsigned int transHistNumBins,
    real_type longDistInvD, real_type transDistInvD, int numSims,
    int thread_histories) {
    int threadX = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadX;

    hiprandStatePhilox4_32_10 local_state{};
    hiprand_init(i, seed, 0, &local_state);

    // Initialise local variables
    real_type trackPosition_x;
    real_type trackPosition_y;
    real_type trackPosition_z;
    real_type trackDirection_x;
    real_type trackDirection_y;
    real_type trackDirection_z;
    real_type stepLength;
    real_type trackLength;
    bool stop;
    unsigned int local_idx;
    real_type rand_val;

    // Perform multiple simulations per thread
    for (int j = 0; j < thread_histories; j++) {
        // Use strides of size blockDim.x*gridDim.x to divide work as
        // equally as possible
        local_idx =
            j * blockDim.x * gridDim.x + blockDim.x * blockIdx.x + threadX;

        if (local_idx >= numSims) {
            break;
        }

        // Initialise local variables
        trackPosition_x = 0.0;
        trackPosition_y = 0.0;
        trackPosition_z = 0.0;
        trackDirection_x = 0.0;
        trackDirection_y = 0.0;
        trackDirection_z = 1.0;

        stepLength = 0.0;
        trackLength = 0.0;
        stop = false;

        // Start the simulation
        while (!stop) {
            // Compute step length
            rand_val = hiprand_uniform(&local_state);

            stepLength = -mean_free_path * log(rand_val);
            if (trackLength > track_limit) {  // Last step, so shorten it
                                              // and stop afterwards
                stepLength = track_limit - trackLength;
                stop = true;
            }

            // Update track positions
            trackPosition_x += stepLength * trackDirection_x;
            trackPosition_y += stepLength * trackDirection_y;
            trackPosition_z += stepLength * trackDirection_z;
            trackLength += stepLength;

            // Update track direction if we are not at the end already
            if (!stop) {
                // Compute new positions based on random collision direction
                rand_val = hiprand_uniform(&local_state);

                real_type cost = sampleCosTheta(screening_param, rand_val);
                real_type sint = sqrt((1.0 - cost) * (1.0 + cost));
                rand_val = hiprand_uniform(&local_state);
                real_type phi = 2.0 * kPI * rand_val;

                real_type u1 = sint * cos(phi);
                real_type u2 = sint * sin(phi);
                real_type u3 = cost;

                // Rotate back to lab frame
                rotateToLabFrame(u1, u2, u3, trackDirection_x, trackDirection_y,
                                 trackDirection_z);

                trackDirection_x = u1;
                trackDirection_y = u2;
                trackDirection_z = u3;
            }
        }

        // Compute longitudinal deviation and its bin index
        real_type longitudinal_deviation = trackPosition_z / trackLength;
        auto longIdx =
            (unsigned int)((longitudinal_deviation + 1.0) * longDistInvD);

        // Compute transversal deviation and its bin index
        real_type transversal_deviation =
            sqrt(trackPosition_x * trackPosition_x +
                 trackPosition_y * trackPosition_y) /
            trackLength;
        auto transIdx = (unsigned int)(transversal_deviation * transDistInvD);

        // Write simulation result to histograms
        atomicAdd(&(longHist[longIdx]), 1);
        atomicAdd(&(transHist[transIdx]), 1);
    }
}

/*
 * Host code
 */

Histograms Simulate(Material material, int numHists) {
    unsigned int *longHist_h, *transHist_h;
    hipError_t cuda_ret;
    const auto seed = std::random_device()();
    // Wake up GPU (seems to be a problem with Peregrine GPUs being in
    // sleep mode, so I wake them up before timing the application)
    // hipFree(0);
    // hipDeviceSynchronize();
    cuda_ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_terminal_positions),
                                      hipFuncCachePreferL1);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    // Initialise space on host for histograms
    longHist_h =
        (unsigned int*)malloc(longiDistNumBin * sizeof(unsigned int));
    transHist_h =
        (unsigned int*)malloc(transDistNumBin * sizeof(unsigned int));

    // Initialise constants
    const real_type theScrPar = computeScrParam(material, thePC2);
    const real_type theMFP = computeMFP(material, theBeta2, theScrPar);
    const real_type theLimit = theMFP * 33.5;

    // Define block and grid dimensions
    constexpr auto num_threads = NUM_THREADS;
    auto num_blocks = div_rounding_up(numHists, num_threads);
    // CUDA has a limit on the grid size
    num_blocks =
        math::min(num_blocks, std::numeric_limits<unsigned int>::max());
    auto thread_histories = div_rounding_up(numHists, num_blocks * num_threads);
    thread_histories = math::max(thread_histories, 1);

    // Initialise histograms on GPU
    unsigned int* longHist_d;
    unsigned int* transHist_d;
    cuda_ret = hipMalloc(&longHist_d, longiDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipMalloc(&transHist_d, transDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise transHist on the GPU.\n");
        exit(-1);
    }

    // Set histogram values to 0
    cuda_ret =
        hipMemset(longHist_d, 0, longiDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set longHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret =
        hipMemset(transHist_d, 0, transDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set transHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    // Run simulation on GPU
    compute_terminal_positions<<<num_blocks, num_threads>>>(
        seed, longHist_d, transHist_d, theScrPar, theMFP, theLimit,
        longiDistNumBin, transDistNumBin, longiDistInvD, transDistInvD,
        numHists, thread_histories);

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to complete simulation kernel.\n");
        exit(-1);
    }
    // Retrieve histograms from GPU
    cuda_ret = hipMemcpy(longHist_h, longHist_d,
                          longiDistNumBin * sizeof(unsigned int),
                          hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve longitudinal histogram from GPU.\n");
        exit(-1);
    }
    cuda_ret = hipMemcpy(transHist_h, transHist_d,
                          transDistNumBin * sizeof(unsigned int),
                          hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    // Copy array pointers into histogram variable
    Histograms histograms{longiDistNumBin, transDistNumBin};
    for (int i = 0; i < longiDistNumBin; ++i) {
        histograms.longiHist[i] = longHist_h[i] * (longiDistInvD / numHists);
    }
    for (int i = 0; i < transDistNumBin; ++i) {
        histograms.transHist[i] = transHist_h[i] * (transDistInvD / numHists);
    }

    // Free buffers
    hipFree(longHist_d);
    hipFree(transHist_d);

    free(longHist_h);
    free(transHist_h);

    return histograms;
}

Histograms Simulate() { return Simulate(GOLD, 1000000); }

}  // namespace Gpu
