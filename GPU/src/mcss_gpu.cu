#include "hip/hip_runtime.h"
#include <cmath>

#include "common.h"
#include "mcss_gpu.cuh"

namespace Gpu {

/*
 * Device code
 */

__device__ real_type sampleCosTheta(real_type scrpar, real_type rn) {
    real_type cost = 1.0 - 2 * scrpar * rn / (1.0 - rn + scrpar);
    return max((real_type)-1.0, min((real_type)1.0, cost));
}

__device__ void rotateToLabFrame(real_type &u, real_type &v, real_type &w,
                                 real_type &u1, real_type &u2, real_type &u3) {
    real_type up = u1 * u1 + u2 * u2;
    if (up > 0.0) {
        up = sqrt(up);
        real_type px = u;
        real_type py = v;
        real_type pz = w;
        u = (u1 * u3 * px - u2 * py) / up + u1 * pz;
        v = (u2 * u3 * px + u1 * py) / up + u2 * pz;
        w = -up * px + u3 * pz;
    } else if (u3 < 0.0) {
        u = -u;
        w = -w;
    }
}

__global__ void compute_terminal_positions(
    hiprandState *states, real_type *longHist, real_type *transHist,
    real_type screening_param, real_type mean_free_path, real_type track_limit,
    const unsigned int longHistNumBins, const unsigned int transHistNumBins,
    real_type longDistInvD, real_type transDistInvD, int numSims) {
    int threadX = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadX;

    // Initialise local histograms
    __shared__ real_type shared_longHist[longiDistNumBin];
    __shared__ real_type shared_transHist[transDistNumBin];

    if (threadX < longHistNumBins) {
        shared_longHist[threadX] = 0.0;
    }
    if (threadX < transHistNumBins) {
        shared_transHist[threadX] = 0.0;
    }

    // Ensure that every thread has initialised the histograms
    __syncthreads();

    // Load cuda random state into local register memory
    hiprandState local_state = states[i];

    // Compute how many simulations each thread will perform
    unsigned int num_sims_per_thread =
        (unsigned int)ceil(float(numSims) / (blockDim.x * gridDim.x));

    // Initialise local variables
    real_type trackPosition_x;
    real_type trackPosition_y;
    real_type trackPosition_z;
    real_type trackDirection_x;
    real_type trackDirection_y;
    real_type trackDirection_z;
    real_type stepLength;
    real_type trackLength;
    bool stop;
    unsigned int local_idx;
    real_type rand_val;

    // Perform multiple simulations per thread
    for (int j = 0; j < num_sims_per_thread; j++) {
        // Use strides of size blockDim.x*gridDim.x to divide work as
        // equally as possible
        local_idx =
            j * blockDim.x * gridDim.x + blockDim.x * blockIdx.x + threadX;

        if (local_idx < numSims) {
            // Initialise local variables
            trackPosition_x = 0.0;
            trackPosition_y = 0.0;
            trackPosition_z = 0.0;
            trackDirection_x = 0.0;
            trackDirection_y = 0.0;
            trackDirection_z = 1.0;

            stepLength = 0.0;
            trackLength = 0.0;
            stop = false;

            // Start the simulation
            while (!stop) {
                // Compute step length
                rand_val = hiprand_uniform(&local_state);

                stepLength = -mean_free_path * log(rand_val);
                if (trackLength > track_limit) {  // Last step, so shorten it
                                                  // and stop afterwards
                    stepLength = track_limit - trackLength;
                    stop = true;
                }

                // Update track positions
                trackPosition_x += stepLength * trackDirection_x;
                trackPosition_y += stepLength * trackDirection_y;
                trackPosition_z += stepLength * trackDirection_z;
                trackLength += stepLength;

                // Update track direction if we are not at the end already
                if (!stop) {
                    // Compute new positions based on random collision direction
                    rand_val = hiprand_uniform(&local_state);

                    real_type cost = sampleCosTheta(screening_param, rand_val);
                    real_type sint = sqrt((1.0 - cost) * (1.0 + cost));
                    rand_val = hiprand_uniform(&local_state);
                    real_type phi = 2.0 * kPI * rand_val;

                    real_type u1 = sint * cos(phi);
                    real_type u2 = sint * sin(phi);
                    real_type u3 = cost;

                    // Rotate back to lab frame
                    rotateToLabFrame(u1, u2, u3, trackDirection_x,
                                     trackDirection_y, trackDirection_z);

                    trackDirection_x = u1;
                    trackDirection_y = u2;
                    trackDirection_z = u3;
                }
            }

            // Compute longitudinal deviation and its bin index
            real_type longitudinal_deviation = trackPosition_z / trackLength;
            unsigned int longIdx =
                (unsigned int)((longitudinal_deviation + 1.0) * longDistInvD);

            // Compute transversal deviation and its bin index
            real_type transversal_deviation =
                sqrt(trackPosition_x * trackPosition_x +
                     trackPosition_y * trackPosition_y) /
                trackLength;
            unsigned int transIdx =
                (unsigned int)(transversal_deviation * transDistInvD);

            // Write simulation result to histograms
            atomicAdd(&(shared_longHist[longIdx]), longDistInvD / numSims);
            atomicAdd(&(shared_transHist[transIdx]), transDistInvD / numSims);
        }
    }

    // Sync up threads after simulations
    __syncthreads();

    // Add shared histogram to global histogram
    if (threadX < longHistNumBins) {
        atomicAdd(&(longHist[threadX]), shared_longHist[threadX]);
    }
    if (threadX < transHistNumBins) {
        atomicAdd(&(transHist[threadX]), shared_transHist[threadX]);
    }
}

__global__ void initialise_RNG(hiprandState *states, int numStates) {
    unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long seed = 0;

    // Use the same seed but different sequence number
    if (i < numStates) {
        hiprand_init(i, seed, 0, &states[i]);
    }
}

/*
 * Host code
 */

Histograms Simulate(Material material, int numHists) {
    real_type *longHist_h, *transHist_h;
    hipError_t cuda_ret;

    // Wake up GPU (seems to be a problem with Peregrine GPUs being in
    // sleep mode, so I wake them up before timing the application)
    // hipFree(0);
    // hipDeviceSynchronize();

    // Initialise space on host for histograms
    longHist_h = (real_type *)malloc(longiDistNumBin * sizeof(real_type));
    transHist_h = (real_type *)malloc(transDistNumBin * sizeof(real_type));

    // Initialise constants
    const real_type theScrPar = computeScrParam(material, thePC2);
    const real_type theMFP = computeMFP(material, theBeta2, theScrPar);
    const real_type theLimit = theMFP * 33.5;

    // Define block and grid dimensions
    int num_blocks = 80;
    dim3 grid_dim, block_dim;
    grid_dim.x = num_blocks;
    block_dim.x = NUM_THREADS;

    // Initialise the RNG state for use in the simulations
    hiprandState *states_d;

    cuda_ret = hipMalloc((void **)&states_d,
                          (num_blocks * NUM_THREADS) * sizeof(hiprandState));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise states on the GPU.\n");
        exit(-1);
    }

    hipDeviceSynchronize();

    initialise_RNG<<<grid_dim, block_dim>>>(states_d, numHists);
    hipDeviceSynchronize();

    // Initialise histograms on GPU
    real_type *longHist_d;
    real_type *transHist_d;
    cuda_ret =
        hipMalloc((void **)&longHist_d, longiDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    cuda_ret =
        hipMalloc((void **)&transHist_d, transDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise transHist on the GPU.\n");
        exit(-1);
    }

    // Set histogram values to 0
    cuda_ret = hipMemset(longHist_d, 0, longiDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set longHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipMemset(transHist_d, 0, transDistNumBin * sizeof(real_type));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set transHist to zero on the GPU.\n");
        exit(-1);
    }

    hipDeviceSynchronize();

    // Run simulation on GPU
    compute_terminal_positions<<<grid_dim, block_dim>>>(
        states_d, longHist_d, transHist_d, theScrPar, theMFP, theLimit,
        longiDistNumBin, transDistNumBin, longiDistInvD, transDistInvD,
        numHists);

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to complete simulation kernel.\n");
        exit(-1);
    }

    // Retrieve histograms from GPU
    cuda_ret =
        hipMemcpy(longHist_h, longHist_d, longiDistNumBin * sizeof(real_type),
                   hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve longitudinal histogram from GPU.\n");
        exit(-1);
    }
    cuda_ret =
        hipMemcpy(transHist_h, transHist_d,
                   transDistNumBin * sizeof(real_type), hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    hipDeviceSynchronize();

    // Copy array pointers into histogram variable
    Histograms histograms{longiDistNumBin, transDistNumBin};
    memcpy(&(histograms.longiHist[0]), &(longHist_h[0]),
           longiDistNumBin * sizeof(real_type));
    memcpy(&(histograms.transHist[0]), &(transHist_h[0]),
           transDistNumBin * sizeof(real_type));

    // Free buffers
    hipFree(longHist_d);
    hipFree(transHist_d);
    hipFree(states_d);

    free(longHist_h);
    free(transHist_h);

    return histograms;
}

Histograms Simulate() { return Simulate(GOLD, 1000000); }

}  // namespace Gpu
