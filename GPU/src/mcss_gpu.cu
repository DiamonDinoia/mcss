#include "hip/hip_runtime.h"
// M. Novak, 9 October 2020 (NOTE: energy is in [MeV] and length in [mm].)
// -----------------------------------------------------------------------
// Modified by R. Brandt 2022
// Modified by M. Barbone 2022#include <cmath>
#include <iostream>
#include <memory>
#include <random>

#include "common.h"
#include "mcss_gpu.h"

namespace Gpu {

static constexpr size_t div_rounding_up(size_t dividend, size_t divisor) {
    return (dividend + divisor - 1) / divisor;
}

__global__ static void compute_terminal_positions(
    unsigned int seed, unsigned int* longHist, unsigned int* transHist,
    const real_type theScrPar, const real_type theMFP, const real_type theLimit,
    const int numSims, const int thread_histories) {
    const int threadX = threadIdx.x;
    const int i       = blockIdx.x * blockDim.x + threadX;

    // Load cuda random state into local register memory
    hiprandStatePhilox4_32_10 local_state{};
    hiprand_init(i, seed, 0, &local_state);
#ifdef USE_SHARED_MEMORY
    __shared__ unsigned int shared_longHist[longiDistNumBin];
    __shared__ unsigned int shared_transHist[transDistNumBin];
    if (threadX < longiDistNumBin) { shared_longHist[threadX] = 0.0; }
    if (threadX < transDistNumBin) { shared_transHist[threadX] = 0.0; }

    // Ensure that every thread has initialised the histograms
    __syncthreads();
#endif
    // Perform multiple simulations per thread
    for (int j = 0; j < thread_histories; j++) {
        // Use strides of size blockDim.x*gridDim.x to divide work as
        // equally as possible

        const auto local_idx =
            j * blockDim.x * gridDim.x + blockDim.x * blockIdx.x + threadX;

        if (local_idx >= numSims) { break; }

        Track     aTrack{};
        real_type trackLength = 0.0;
        bool      stop        = false;
        do {
            real_type stepLength =
                -theMFP * std::log(hiprand_uniform(&local_state));
            trackLength += stepLength;
            if (trackLength > theLimit) {
                stepLength = theLimit - aTrack.fTrackLength;
                stop       = true;
            }
            aTrack.fPosition[0] += aTrack.fDirection[0] * stepLength;
            aTrack.fPosition[1] += aTrack.fDirection[1] * stepLength;
            aTrack.fPosition[2] += aTrack.fDirection[2] * stepLength;
            aTrack.fTrackLength += stepLength;
            if (!stop) {
                const real_type cost =
                    sampleCosTheta(theScrPar, hiprand_uniform(&local_state));
                const real_type dum0 = 1.0 - cost;
                const real_type sint = std::sqrt(dum0 * (2.0 - dum0));
                const real_type phi  = 2.0 * kPI * hiprand_uniform(&local_state);
                real_type       u1   = sint * std::cos(phi);
                real_type       u2   = sint * std::sin(phi);
                real_type       u3   = cost;
                rotateToLabFrame(u1, u2, u3, aTrack.fDirection[0],
                                 aTrack.fDirection[1], aTrack.fDirection[2]);
                aTrack.fDirection[0] = u1;
                aTrack.fDirection[1] = u2;
                aTrack.fDirection[2] = u3;
            }
        } while (!stop);

        // Calculate final longitudinal location.
        const real_type longi = aTrack.fPosition[2] / aTrack.fTrackLength;
        const unsigned  lIndx = (longi + 1.0) * longiDistInvD;
        // Increment the relevant element of the thread-specific
        // longitudinal histogram.
        const real_type trans =
            std::sqrt(aTrack.fPosition[0] * aTrack.fPosition[0] +
                      aTrack.fPosition[1] * aTrack.fPosition[1]) /
            aTrack.fTrackLength;
        const unsigned tIndx = trans * transDistInvD;
        // Increment the relevent element of the thread-specific
        // trans histogram.
#ifndef USE_SHARED_MEMORY
        atomicAdd(&(longHist[lIndx]), 1);
        atomicAdd(&(transHist[tIndx]), 1);
#endif
#ifdef USE_SHARED_MEMORY

        atomicAdd(&(shared_longHist[lIndx]), 1);
        atomicAdd(&(shared_transHist[tIndx]), 1);
#endif
    }
#ifdef USE_SHARED_MEMORY
    // Add shared histogram to global histogram
    if (threadX < longiDistNumBin) {
        atomicAdd(&(longHist[threadX]), shared_longHist[threadX]);
    }
    if (threadX < transDistNumBin) {
        atomicAdd(&(longHist[threadX]), shared_transHist[threadX]);
    }
#endif
}

/*
 * Host code
 */

Histograms Simulate(Material material, int numHists) {
    hipError_t cuda_ret;
    const auto  seed = std::random_device()();
    // Wake up GPU (seems to be a problem with Peregrine GPUs being in
    // sleep mode, so I wake them up before timing the application)
    // hipFree(0);
    // hipDeviceSynchronize();
#ifndef USE_SHARED_MEMORY
    cuda_ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_terminal_positions),
                                      hipFuncCachePreferL1);
#else
    cuda_ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_terminal_positions),
                                      hipFuncCachePreferShared);
#endif
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    // Initialise space on host for histograms
    auto longHist_h  = std::make_unique<unsigned int[]>(longiDistNumBin);
    auto transHist_h = std::make_unique<unsigned int[]>(transDistNumBin);

    // Initialise constants
    const real_type theScrPar = computeScrParam(material, thePC2);
    const real_type theMFP    = computeMFP(material, theBeta2, theScrPar);
    const real_type theLimit  = theMFP * 33.5;

    // Define block and grid dimensions
    constexpr auto num_threads = NUM_THREADS;
    auto           num_blocks  = div_rounding_up(numHists, num_threads);
    // CUDA has a limit on the grid size
    num_blocks            = math::min(num_blocks, GRID_SIZE);

    auto thread_histories = div_rounding_up(numHists, num_blocks * num_threads);
    thread_histories      = math::max(thread_histories, 1);

    // Initialise histograms on GPU
    unsigned int* longHist_d;
    unsigned int* transHist_d;
    cuda_ret = hipMalloc(&longHist_d, longiDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise longHist on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipMalloc(&transHist_d, transDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to initialise transHist on the GPU.\n");
        exit(-1);
    }

    // Set histogram values to 0
    cuda_ret =
        hipMemset(longHist_d, 0, longiDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set longHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret =
        hipMemset(transHist_d, 0, transDistNumBin * sizeof(unsigned int));
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to set transHist to zero on the GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    // Run simulation on GPU
    compute_terminal_positions<<<num_blocks, num_threads>>>(
        seed, longHist_d, transHist_d, theScrPar, theMFP, theLimit, numHists,
        thread_histories);

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to complete simulation kernel.\n");
        exit(-1);
    }
    // Retrieve histograms from GPU
    cuda_ret = hipMemcpy(longHist_h.get(), longHist_d,
                          longiDistNumBin * sizeof(unsigned int),
                          hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve longitudinal histogram from GPU.\n");
        exit(-1);
    }
    cuda_ret = hipMemcpy(transHist_h.get(), transHist_d,
                          transDistNumBin * sizeof(unsigned int),
                          hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("ERROR: Failed to retrieve transversal histogram from GPU.\n");
        exit(-1);
    }

    // Copy array pointers into histogram variable
    Histograms histograms{longiDistNumBin, transDistNumBin};
    for (int i = 0; i < longiDistNumBin; ++i) {
        histograms.longiHist[i] = longHist_h[i] * (longiDistInvD / numHists);
    }
    for (int i = 0; i < transDistNumBin; ++i) {
        histograms.transHist[i] = transHist_h[i] * (transDistInvD / numHists);
    }

    // Free buffers
    hipFree(longHist_d);
    hipFree(transHist_d);

    return histograms;
}

Histograms Simulate() { return Simulate(GOLD, 1000000); }

}  // namespace Gpu
